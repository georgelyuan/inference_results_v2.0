/*
 * Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdint.h>

struct UNet3DParams
{
    int image_d;
    int image_h;
    int image_w;
    int image_size;
    int offset_d;
    int offset_h;
    int offset_w;
    int roi_dhw;
    int roi_size;
    int in_ch;
    int out_ch;

    UNet3DParams()
    {
        image_d = 256;
        image_h = 256;
        image_w = 256;
        image_size = 256 * 256 * 256;
        offset_d = 0;
        offset_h = 0;
        offset_w = 0;
        roi_dhw = 128;
        roi_size = 128 * 128 * 128;
        in_ch = 1;
        out_ch = 3;
    }
};

extern "C" __global__ void UNet3DKiTS19SliceKernelFP32Linear(
    const float* __restrict__ d_in, float* __restrict__ d_out, const UNet3DParams* p)
{
    const int d = blockIdx.x;
    const int h = blockIdx.y;
    const int w = threadIdx.x;

    d_out[p->roi_dhw * p->roi_dhw * d + p->roi_dhw * h + w]
        = d_in[p->image_w * p->image_h * (p->offset_d + d) + p->image_w * (p->offset_h + h) + (p->offset_w + w)];
}

extern "C" __global__ void UNet3DKiTS19SliceKernelI8Linear(
    const int8_t* __restrict__ d_in, int8_t* __restrict__ d_out, const UNet3DParams* p)
{
    int d = blockIdx.x;
    int h = blockIdx.y;
    int w = threadIdx.x;

    d_out[p->roi_dhw * p->roi_dhw * d + p->roi_dhw * h + w]
        = d_in[p->image_w * p->image_h * (p->offset_d + d) + p->image_w * (p->offset_h + h) + (p->offset_w + w)];
}

extern "C" __global__ void UNet3DKiTS19SliceKernelI8CDHW32(
    const int8_t* __restrict__ d_in, int8_t* __restrict__ d_out, const UNet3DParams* p)
{
    int d = blockIdx.x;
    int h = blockIdx.y;
    int w = threadIdx.x;

    d_out[32 * (p->roi_dhw * p->roi_dhw * d + p->roi_dhw * h + w)]
        = d_in[p->image_w * p->image_h * (p->offset_d + d) + p->image_w * (p->offset_h + h) + (p->offset_w + w)];
}

extern "C" __global__ void UNet3DKiTS19PatchKernel(const __half* __restrict__ d_in, const __half* __restrict__ d_patch,
    __half* __restrict__ d_out, const UNet3DParams* p)
{
    int d = blockIdx.x;
    int h = blockIdx.y;
    int w = threadIdx.x;

#pragma unroll
    for (int c = 0; c < p->out_ch; ++c)
    {
        d_out[p->image_h * p->image_w * (p->offset_d + d) + p->image_w * (p->offset_h + h) + (p->offset_w + w)
            + p->image_size * c]
            += d_in[p->roi_dhw * p->roi_dhw * d + p->roi_dhw * h + w + p->roi_size * c]
            * d_patch[p->roi_dhw * p->roi_dhw * d + p->roi_dhw * h + w];
    }
}

extern "C" __global__ void UNet3DKiTS19ArgMaxKernel(
    const __half* __restrict__ d_in, int8_t* __restrict__ d_out, const UNet3DParams* p)
{
    int d = blockIdx.x;
    int h = blockIdx.y;
    int w = threadIdx.x;

    __half a = d_in[p->image_h * p->image_w * d + p->image_w * h + w];
    __half b = d_in[p->image_h * p->image_w * d + p->image_w * h + w + p->image_size];
    __half c = d_in[p->image_h * p->image_w * d + p->image_w * h + w + 2 * p->image_size];
    __half m = b;
    uint8_t l = 1;
    if (a > b)
    {
        m = a;
        l = 0;
    }
    d_out[p->image_h * p->image_w * d + p->image_w * h + w] = m > c ? l : 2;
}
