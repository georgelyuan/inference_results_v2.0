#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "unet3d_sw.cuh"

namespace lwis {

__global__ void UNet3DKiTS19SliceKernelFP32Linear(const float* __restrict__ d_in,
                                                  float* __restrict__ d_out, const UNet3DParams p)
{
    const int d = blockIdx.x;
    const int h = blockIdx.y;
    const int w = threadIdx.x;

    if(d < p.roi_dhw && h < p.roi_dhw && w < p.roi_dhw)
    {
        d_out[p.roi_dhw * p.roi_dhw * d + p.roi_dhw * h + w] =
            d_in[p.image_w * p.image_h * (p.offset_d + d) + p.image_w * (p.offset_h + h) +
                 (p.offset_w + w)];
    }
}

__global__ void UNet3DKiTS19SliceKernelI8Linear(const int8_t* __restrict__ d_in,
                                                int8_t* __restrict__ d_out, const UNet3DParams p)
{
    const int d = blockIdx.x;
    const int h = blockIdx.y;
    const int w = threadIdx.x;

    if(d < p.roi_dhw && h < p.roi_dhw && w < p.roi_dhw)
    {
        d_out[p.roi_dhw * p.roi_dhw * d + p.roi_dhw * h + w] =
            d_in[p.image_w * p.image_h * (p.offset_d + d) + p.image_w * (p.offset_h + h) +
                 (p.offset_w + w)];
    }
}

__global__ void UNet3DKiTS19SliceKernelI8CDHW32(const int8_t* __restrict__ d_in,
                                                int8_t* __restrict__ d_out, const UNet3DParams p)
{
    const int d = blockIdx.x;
    const int h = blockIdx.y;
    const int w = threadIdx.x;

    if(d < p.roi_dhw && h < p.roi_dhw && w < p.roi_dhw)
    {
        d_out[32 * (p.roi_dhw * p.roi_dhw * d + p.roi_dhw * h + w)] =
            d_in[p.image_w * p.image_h * (p.offset_d + d) + p.image_w * (p.offset_h + h) +
                 (p.offset_w + w)];
    }
}

__global__ void UNet3DKiTS19PatchKernel(const __half* __restrict__ d_in,
                                        const __half* __restrict__ d_patch,
                                        __half* __restrict__ d_out, const UNet3DParams p)
{
    const int d = blockIdx.x;
    const int h = blockIdx.y;
    const int w = threadIdx.x;

    if(d < p.roi_dhw && h < p.roi_dhw && w < p.roi_dhw)
    {
#pragma unroll
        for(int c = 0; c < p.out_ch; ++c)
        {
            d_out[p.image_h * p.image_w * (p.offset_d + d) + p.image_w * (p.offset_h + h) +
                  (p.offset_w + w) + p.image_size * c] +=
                d_in[p.roi_dhw * p.roi_dhw * d + p.roi_dhw * h + w + p.roi_size * c] *
                d_patch[p.roi_dhw * p.roi_dhw * d + p.roi_dhw * h + w];
        }
    }
}

__global__ void UNet3DKiTS19ArgMaxKernel(const __half* __restrict__ d_in,
                                         int8_t* __restrict__ d_out, const UNet3DParams p)
{
    const int d = blockIdx.x;
    const int h = blockIdx.y;
    const int w = threadIdx.x;

    __half a = d_in[p.image_h * p.image_w * d + p.image_w * h + w];
    __half b = d_in[p.image_h * p.image_w * d + p.image_w * h + w + p.image_size];
    __half c = d_in[p.image_h * p.image_w * d + p.image_w * h + w + 2 * p.image_size];
    __half m = b;
    uint8_t l = 1;
    if(a > b)
    {
        m = a;
        l = 0;
    }
    if(d < p.image_d && h < p.image_h && w < p.image_w)
    {
        d_out[p.image_h * p.image_w * d + p.image_w * h + w] = m > c ? l : 2;
    }
}

void UNet3DKiTS19SliceKernelFP32Linear_wrapper(void* d_in, void* d_out, const UNet3DParams* p,
                                               const hipStream_t stream = 0,
                                               const int deviceId = 0)
{
    hipSetDevice(deviceId);
    // for slicing
    dim3 dimBlock_slice(p->roi_dhw, 1, 1);
    dim3 dimGrid_slice(p->roi_dhw, p->roi_dhw, 1);
    UNet3DKiTS19SliceKernelFP32Linear<<<dimGrid_slice, dimBlock_slice, 0, stream>>>(
        static_cast<float*>(d_in), static_cast<float*>(d_out), *p);
}

void UNet3DKiTS19SliceKernelI8Linear_wrapper(void* d_in, void* d_out, const UNet3DParams* p,
                                             const hipStream_t stream = 0, const int deviceId = 0)
{
    hipSetDevice(deviceId);
    // for slicing
    dim3 dimBlock_slice(p->roi_dhw, 1, 1);
    dim3 dimGrid_slice(p->roi_dhw, p->roi_dhw, 1);
    UNet3DKiTS19SliceKernelI8Linear<<<dimGrid_slice, dimBlock_slice, 0, stream>>>(
        static_cast<int8_t*>(d_in), static_cast<int8_t*>(d_out), *p);
}

void UNet3DKiTS19SliceKernelI8CDHW32_wrapper(void* d_in, void* d_out, const UNet3DParams* p,
                                             const hipStream_t stream = 0, const int deviceId = 0)
{
    hipSetDevice(deviceId);
    // for slicing
    dim3 dimBlock_slice(p->roi_dhw, 1, 1);
    dim3 dimGrid_slice(p->roi_dhw, p->roi_dhw, 1);
    UNet3DKiTS19SliceKernelI8CDHW32<<<dimGrid_slice, dimBlock_slice, 0, stream>>>(
        static_cast<int8_t*>(d_in), static_cast<int8_t*>(d_out), *p);
}

void UNet3DKiTS19PatchKernel_wrapper(void* d_in, void* d_patch, void* d_out, const UNet3DParams* p,
                                     const hipStream_t stream = 0, const int deviceId = 0)
{
    hipSetDevice(deviceId);
    // for Gaussian patching & accumulating
    dim3 dimBlock_patch(p->roi_dhw, 1, 1);
    dim3 dimGrid_patch(p->roi_dhw, p->roi_dhw, 1);
    UNet3DKiTS19PatchKernel<<<dimGrid_patch, dimBlock_patch, 0, stream>>>(
        static_cast<__half*>(d_in), static_cast<__half*>(d_patch), static_cast<__half*>(d_out), *p);
}

void UNet3DKiTS19ArgMaxKernel_wrapper(void* d_in, void* d_out, const UNet3DParams* p,
                                      const hipStream_t stream = 0, const int deviceId = 0)
{
    hipSetDevice(deviceId);
    // for final ArgMax
    dim3 dimBlock_argmax(p->image_w, 1, 1);
    dim3 dimGrid_argmax(p->image_d, p->image_h, 1);
    UNet3DKiTS19ArgMaxKernel<<<dimGrid_argmax, dimBlock_argmax, 0, stream>>>(
        static_cast<__half*>(d_in), static_cast<int8_t*>(d_out), *p);
}

} // namespace lwis